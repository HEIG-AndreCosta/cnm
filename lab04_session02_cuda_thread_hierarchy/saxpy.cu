#include "hip/hip_runtime.h"
// SAXPY Single precision A * X plus Y

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// TODO: CUDA kernel for scalar multiplication and vector addition, pay attention to stride

__global__ void scalar_multiplication(size_t n, float a, float *x, float *y,
				      float *z)
{
	size_t index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < n) {
		z[index] = a * x[index] + y[index];
	}
}
// Scalar multiplication and vector addition
void saxpy_cpu(int n, float a, float *x, float *y, float *z)
{
	for (int i = 0; i < n; ++i) {
		z[i] = a * x[i] + y[i];
	}
}

// Check saxpy result
bool check_saxpy(int n, float a, const float *x, const float *y, float *z)
{
	for (int i = 0; i < n; ++i) {
		if (z[i] != a * x[i] + y[i])
			return false;
	}
	return true;
}

int main(int argc, char const *argv[])
{


    if (argc < 3 || argc > 4) {
        printf("Usage: %s <num_block> <block_size>   \n", argv[0]);
        printf("Usage: %s <num_block_x> <num_block_y> <block_size>   \n", argv[0]);
        return 1;
    }

    const int block_size = atoi(argv[2]);
    const int num_block = atoi(argv[1]);

    if (block_size <= 0 || num_block <= 0) {
        printf("Block size or num block must be positive integers.\n");
        return 1;
    }

    const int num_block_y = argc == 4 ? atoi(argv[2]):0;
    
    if (num_block_y <= 0) {
        printf("Block size must be positive integers.\n");
        return 1;
    }

    if (argc == 3) {
        printf("Block size: %d\n", block_size);
        printf("Num block: %d\n", num_block);
    }
    else {
        printf("Block size: %d\n", block_size);
        printf("Num block x: %d\n", num_block);
        printf("Num block y: %d\n", num_block_y);
    }

	int n = 1 << 20; //2^20
	float a = 2;

	// Allocate host memory
	float *h_x = (float *)malloc(n * sizeof(float));
	float *h_y = (float *)malloc(n * sizeof(float));
	float *h_z = (float *)malloc(n * sizeof(float));

	if (h_x == NULL || h_y == NULL || h_z == NULL) {
		printf("Error allocating host memory");
		exit(EXIT_FAILURE);
	}

	// Initialize input vectores
	for (int i = 0; i < n; ++i) {
		h_x[i] = 1.0;
		h_y[i] = (float)(i);
	}

	printf("Running SAXPY in CPU...\n");
	saxpy_cpu(n, a, h_x, h_y, h_z);

	//Check SAXPY CPU results
	printf("Checking CPU SAXPY: %s\n",
	       check_saxpy(n, a, h_x, h_y, h_z) ? "Success" : "Error");

	/* ********************************************************************* */

	const size_t size = n * sizeof(float);
	float *d_x, *d_y, *d_z;
	hipError_t error = hipMalloc((void **)&d_x, size);
	if (error != hipSuccess) {
		return 1;
	}
	error = hipMalloc((void **)&d_y, size);
	if (error != hipSuccess) {
		hipFree(d_x);
		return 1;
	}
	error = hipMalloc((void **)&d_z, size);
	if (error != hipSuccess) {
		hipFree(d_x);
		hipFree(d_y);
		return 1;
	}

	//TODO: Copy memory from host to device and check for errors

	error = hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		hipFree(d_x);
		hipFree(d_y);
		hipFree(d_z);
		return 1;
	}
	error = hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		hipFree(d_x);
		hipFree(d_y);
		hipFree(d_z);
		return 1;
	}
	error = hipMemcpy(d_z, h_z, size, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		hipFree(d_x);
		hipFree(d_y);
		hipFree(d_z);
		return 1;
	}

	//TODO: Call kernel and check for errors
    if (argc == 3 ) {
        scalar_multiplication<<<num_block, block_size>>>(n, a, d_x, d_y,
                                    d_z);
    }
    else {
        scalar_multiplication<<< {num_block, num_block_y}, block_size>>>(n, a, d_x, d_y,
                                    d_z);
    }
        
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error after kernel: %s\n", hipGetErrorString(err));
        hipFree(d_x);
		hipFree(d_y);
		hipFree(d_z);
        return 1;
    }

	//TODO: Copy memory from device to host and check for errors

	error = hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		hipFree(d_x);
		hipFree(d_y);
		hipFree(d_z);
		return 1;
	}

	//TODO: Free memory from device to host and check for errors

	error = hipFree(d_x);
	if (error != hipSuccess) {
		printf("Couldn't free dx\n");
	}
	error = hipFree(d_y);
	if (error != hipSuccess) {
		printf("Couldn't free dy\n");
	}
	error = hipFree(d_z);
	if (error != hipSuccess) {
		printf("Couldn't free dz\n");
	}
	/* ********************************************************************* */

	//Check SAXPY GPU results
	printf("Checking GPU SAXPY: %s\n",
	       check_saxpy(n, a, h_x, h_y, h_z) ? "Success" : "Error");

	free(h_x);
	free(h_y);
	free(h_z);

	return 0;
}

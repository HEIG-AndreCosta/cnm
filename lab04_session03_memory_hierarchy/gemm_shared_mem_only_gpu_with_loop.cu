#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define THRESHOLD  0.00001

// a: m x n
// b: n x p
// c: m x p
__global__ void gemm_shared_mem(float *a, float *b, float *c, size_t m,
				size_t n, size_t p, size_t tile_size_x,
				size_t tile_size_y)
{
	// row a column based on thread index inside the block and grid
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float sum = 0.0;

	const int TILE_SIZE = blockDim.x;

	// Shared memory for tiles
	__shared__ float aTile
		[tile_size_x]
		[tile_size_y]; // TILE_SIZE must not exceed shared memory limits
	__shared__ float bTile[tile_size_x][tile_size_y];

	// Iterate over tiles in the shared dimension
	for (int t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; ++t) {
		// Load elements of the current tile from global memory into shared memory
		if (row < m && t * TILE_SIZE + threadIdx.x < n) {
			aTile[threadIdx.y][threadIdx.x] =
				a[row * n + t * TILE_SIZE + threadIdx.x];
		} else {
			aTile[threadIdx.y][threadIdx.x] = 0.0f;
		}

		if (col < p && t * TILE_SIZE + threadIdx.y < n) {
			bTile[threadIdx.y][threadIdx.x] =
				b[(t * TILE_SIZE + threadIdx.y) * p + col];
		} else {
			bTile[threadIdx.y][threadIdx.x] = 0.0f;
		}
		// Synchronize to ensure all threads have loaded their tiles
		__syncthreads();

		// Multiply the tiles
		for (int k = 0; k < TILE_SIZE; ++k) {
			sum += aTile[threadIdx.y][k] * bTile[k][threadIdx.x];
		}
		// Synchronize to ensure the tiles aren't overwritten
		__syncthreads();
	}

	c[row * m + col] += sum;
}

// CPU Matrix multiplication
void gemm_cpu(float const *a, float const *b, float *c, size_t m, size_t n,
	      size_t p)
{
	// Iterate over rows in matrix a
	for (size_t i = 0; i < m; ++i) {
		// Iterate over columns in matrix b
		for (size_t j = 0; j < p; ++j) {
			float acc_sum = 0;
			// Iterate over each elment in row and column
			for (size_t k = 0; k < n; ++k) {
				acc_sum += a[i * n + k] * b[k * p + j];
			}
			c[i * p + j] = acc_sum;
		}
	}
}

// Check gemm result
bool check_gemm(float const *a, float const *b, float const *c, size_t m,
		size_t n, size_t p)
{
	for (size_t i = 0; i < m; ++i) {
		for (size_t j = 0; j < p; ++j) {
			float acc_sum = 0;
			for (size_t k = 0; k < n; ++k) {
				acc_sum += a[i * n + k] * b[k * p + j];
			}
			if (abs(c[i * p + j] - acc_sum) > THRESHOLD) {
				return false;
			}
		}
	}
	return true;
}

int main(int argc, char const *argv[])
{
	size_t = global_size = 512;
	size_t m = global_size;
	size_t n = global_size;
	size_t p = global_size;

	float event_elaspsed_time_ms = 0;

	// a: m x n
	// b: n x p
	// c: m x p

	// Allocate host memory
	float *h_a = new float[m * n];
	float *h_b = new float[n * p];
	float *h_c = new float[m * p];

	if (h_a == NULL || h_b == NULL || h_c == NULL) {
		printf("Error allocating host memory");
		exit(EXIT_FAILURE);
	}

	// Initialize input matrix
	for (size_t i = 0; i < m * n; ++i) {
		h_a[i] = 1.0;
	};

	for (size_t i = 0; i < n * p; ++i) {
		h_b[i] = 1.0;
	};

	// Clean up result
	for (size_t i = 0; i < m * p; ++i) {
		h_c[i] = 0.0;
	};

	/* ********************************************************************* */

	for (int i = 0; i < global_size; i += 2) {
		float *d_a, *d_b, *d_c;

		hipEvent_t start, stop;

		hipEventCreate(&start);
		hipEventCreate(&stop);

		// Device memory allocation
		hipMalloc((void **)&d_a, m * n * sizeof(float));
		hipMalloc((void **)&d_b, n * p * sizeof(float));
		hipMalloc((void **)&d_c, m * p * sizeof(float));

		// Device memory inialization
		hipMemcpy(d_a, h_a, m * n * sizeof(float),
			   hipMemcpyHostToDevice);
		hipMemcpy(d_b, h_b, n * p * sizeof(float),
			   hipMemcpyHostToDevice);

		// Excution configuration
		dim3 grid_size(m / BLOCK_SIZE, p / BLOCK_SIZE);
		dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);

		// Kernel execution and measuring
		printf("Running GEMM shared mem in GPU...\n");
		hipEventRecord(start);
		gemm_shared_mem<<<grid_size, block_size> > >(d_a, d_b, d_c, m,
							     n, p, i, i);
		hipEventRecord(stop);

		// Copy result from device
		hipMemcpy(h_c, d_c, m * p * sizeof(float),
			   hipMemcpyDeviceToHost);

		hipEventElapsedTime(&event_elaspsed_time_ms, start, stop);

		printf("%d,%s, %.3f ms\n", i,
		       check_gemm(h_a, h_b, h_c, m, n, p) ? "Success" : "Error",
		       event_elaspsed_time_ms);

		// Event cleaning
		hipEventDestroy(start);
		hipEventDestroy(stop);

		// Memory deallocation
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);

		delete[] h_a;
		delete[] h_b;
		delete[] h_c;
	}

	return 0;
}

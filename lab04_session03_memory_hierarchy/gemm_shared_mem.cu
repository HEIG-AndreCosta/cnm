#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16
#define THRESHOLD  0.00001

// a: m x n
// b: n x p
// c: m x p
__global__ void gemm_shared_mem(float *a, float *b, float *c, size_t m,
				size_t n, size_t p)
{
	// row a column based on thread index inside the block and grid
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float sum = 0.0;

	const int TILE_SIZE = blockDim.x;

	// Shared memory for tiles
	__shared__ float aTile
		[BLOCK_SIZE]
		[BLOCK_SIZE]; // TILE_SIZE must not exceed shared memory limits
	__shared__ float bTile[BLOCK_SIZE][BLOCK_SIZE];

	// Iterate over tiles in the shared dimension
	for (int t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; ++t) {
		// Load elements of the current tile from global memory into shared memory
		if (row < m && t * TILE_SIZE + threadIdx.x < n) {
			aTile[threadIdx.y][threadIdx.x] =
				a[row * n + t * TILE_SIZE + threadIdx.x];
		} else {
			aTile[threadIdx.y][threadIdx.x] = 0.0f;
		}

		if (col < p && t * TILE_SIZE + threadIdx.y < n) {
			bTile[threadIdx.y][threadIdx.x] =
				b[(t * TILE_SIZE + threadIdx.y) * p + col];
		} else {
			bTile[threadIdx.y][threadIdx.x] = 0.0f;
		}
		// Synchronize to ensure all threads have loaded their tiles
		__syncthreads();

		// Multiply the tiles
		for (int k = 0; k < TILE_SIZE; ++k) {
			sum += aTile[threadIdx.y][k] * bTile[k][threadIdx.x];
		}
		// Synchronize to ensure the tiles aren't overwritten
		__syncthreads();
	}

	c[row * m + col] += sum;
}

// CPU Matrix multiplication
void gemm_cpu(float const *a, float const *b, float *c, size_t m, size_t n,
	      size_t p)
{
	// Iterate over rows in matrix a
	for (size_t i = 0; i < m; ++i) {
		// Iterate over columns in matrix b
		for (size_t j = 0; j < p; ++j) {
			float acc_sum = 0;
			// Iterate over each elment in row and column
			for (size_t k = 0; k < n; ++k) {
				acc_sum += a[i * n + k] * b[k * p + j];
			}
			c[i * p + j] = acc_sum;
		}
	}
}

// Check gemm result
bool check_gemm(float const *a, float const *b, float const *c, size_t m,
		size_t n, size_t p)
{
	for (size_t i = 0; i < m; ++i) {
		for (size_t j = 0; j < p; ++j) {
			float acc_sum = 0;
			for (size_t k = 0; k < n; ++k) {
				acc_sum += a[i * n + k] * b[k * p + j];
			}
			if (abs(c[i * p + j] - acc_sum) > THRESHOLD) {
				return false;
			}
		}
	}
	return true;
}

int main(int argc, char const *argv[])
{
	size_t m = 1024;
	size_t n = 1024;
	size_t p = 1024;

	float event_elaspsed_time_ms = 0;

	// a: m x n
	// b: n x p
	// c: m x p

	// Allocate host memory
	float *h_a = new float[m * n];
	float *h_b = new float[n * p];
	float *h_c = new float[m * p];

	if (h_a == NULL || h_b == NULL || h_c == NULL) {
		printf("Error allocating host memory");
		exit(EXIT_FAILURE);
	}

	// Initialize input matrix
	for (size_t i = 0; i < m * n; ++i) {
		h_a[i] = 1.0;
	};

	for (size_t i = 0; i < n * p; ++i) {
		h_b[i] = 1.0;
	};

	printf("Running GEMM in CPU...\n");
	auto cpu_start = std::chrono::high_resolution_clock::now();
	gemm_cpu(h_a, h_b, h_c, m, n, p);
	auto cpu_stop = std::chrono::high_resolution_clock::now();

	// Check SAXPY CPU results
	printf("Checking CPU GEMM: %s\n",
	       check_gemm(h_a, h_b, h_c, m, n, p) ? "Success" : "Error");

	event_elaspsed_time_ms =
		std::chrono::duration<float, std::milli>(cpu_stop - cpu_start)
			.count();

	printf("Finished GEMM in CPU in %.3f ms\n", event_elaspsed_time_ms);

	// Clean up result
	for (size_t i = 0; i < m * p; ++i) {
		h_c[i] = 0.0;
	};

	/* ********************************************************************* */

	float *d_a, *d_b, *d_c;

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Device memory allocation
	hipMalloc((void **)&d_a, m * n * sizeof(float));
	hipMalloc((void **)&d_b, n * p * sizeof(float));
	hipMalloc((void **)&d_c, m * p * sizeof(float));

	// Device memory inialization
	hipMemcpy(d_a, h_a, m * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, n * p * sizeof(float), hipMemcpyHostToDevice);

	// Excution configuration
	dim3 grid_size(m / BLOCK_SIZE, p / BLOCK_SIZE);
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);

	// Kernel execution and measuring
	printf("Running GEMM shared mem in GPU...\n");
	hipEventRecord(start);
	gemm_shared_mem<<<grid_size, block_size> > >(d_a, d_b, d_c, m, n, p);
	hipEventRecord(stop);

	// Copy result from device
	hipMemcpy(h_c, d_c, m * p * sizeof(float), hipMemcpyDeviceToHost);

	// Check result
	printf("Checking GPU shared mem GEMM: %s\n",
	       check_gemm(h_a, h_b, h_c, m, n, p) ? "Success" : "Error");

	hipEventElapsedTime(&event_elaspsed_time_ms, start, stop);

	printf("Finished GEMM with shared mem in GPU in %.3f ms\n",
	       event_elaspsed_time_ms);

	// Event cleaning
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Memory deallocation
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	delete[] h_a;
	delete[] h_b;
	delete[] h_c;

	return 0;
}

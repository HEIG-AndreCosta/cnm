#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 10000000
#define BLOCK_WIDTH 1000

__global__ void increment(int *a, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	i = i % size;
	atomicInc(a + i, 1);
}

// TODO implement increment kernel with CUDA atomics

int main(int argc, char const *argv[])
{
	size_t size = 10;

	int *h_a = new int[size];
	int *d_a;

	float event_elaspsed_time_ms = 0.0;

	/* ********************************************************************* */

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void **)&d_a, size * sizeof(int));
	hipMemset((void *)d_a, 0, size * sizeof(int));

	hipEventRecord(start);
	increment<<<std::ceil(NUM_THREADS / BLOCK_WIDTH), BLOCK_WIDTH>>>(
		d_a, size);
	hipEventRecord(stop);

	hipMemcpy(h_a, d_a, size * sizeof(int), hipMemcpyDeviceToHost);

	hipEventElapsedTime(&event_elaspsed_time_ms, start, stop);

	hipFree(d_a);

	/* ********************************************************************* */

	printf("{");
	for (size_t i = 0; i < size; ++i)
		printf(" %d", h_a[i]);
	printf(" }\n");

	printf("%g ms\n", event_elaspsed_time_ms);

	delete[] h_a;

	return 0;
}
